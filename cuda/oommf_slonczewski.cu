#include "hip/hip_runtime.h"
// Original implementation by Mykola Dvornik for mumax2
// Modified for mumax3 by Arne Vansteenkiste, 2013

#include <stdint.h>
#include "float3.h"
#include "constants.h"

extern "C" __global__ void
addoommfslonczewskitorque(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                     float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz, float* __restrict__ jz,
                     float* __restrict__ pxLUT, float* __restrict__ pyLUT, float* __restrict__ pzLUT,
                     float* __restrict__ msatLUT, float* __restrict__ alphaLUT, float flt,
                     float* __restrict__ pfixLUT, float* __restrict__ pfreeLUT,
                     float* __restrict__ lambdafixLUT, float* __restrict__ lambdafreeLUT,
		     float* __restrict__ epsilonPrimeLUT,
                     uint8_t* __restrict__ regions, int N) {

	int I =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (I < N) {

		float3 m = make_float3(mx[I], my[I], mz[I]);
		float  J = jz[I];

		// read parameters
		uint8_t region       = regions[I];

		float3 p            = normalized(make_float3(pxLUT[region], pyLUT[region], pzLUT[region]));
		float  Ms           = msatLUT[region];
		float  alpha        = alphaLUT[region];
		float  pfix         = pfixLUT[region];
		float  pfree        = pfreeLUT[region];
		float  lambdafix    = lambdafixLUT[region];
		float  lambdafree   = lambdafreeLUT[region];
		float  epsilonPrime = epsilonPrimeLUT[region];

		if (J == 0.0f || Ms == 0.0f) {
			return;
		}

		float beta    = (HBAR / QE) * (J / (2.0f *flt*Ms) );
		float lambdafix2 = lambdafix * lambdafix;
		float lambdafree2 = lambdafree * lambdafree;
		float lambdafreePlus = sqrt(lambdafree2 + 1.0f);
		float lambdafixPlus = sqrt(lambdafix2 + 1.0f);
		float lambdafreeMinus = sqrt(lambdafree2 - 1.0f);
		float lambdafixMinus = sqrt(lambdafix2 - 1.0f);
		float plus_ratio = lambdafreePlus / lambdafixPlus;
		float minus_ratio = 1.0f;
		if (lambdafreeMinus > 0) {
		   	minus_ratio = lambdafixMinus / lambdafreeMinus;
		}
		// Compute q_plus and q_minus
		float plus_factor = pfix * lambdafix2 * plus_ratio;
		float minus_factor = pfree * lambdafree2 * minus_ratio;
		float q_plus = plus_factor + minus_factor;
		float q_minus = plus_factor - minus_factor;
		float lplus2 = lambdafreePlus * lambdafixPlus;
		float lminus2 = lambdafreeMinus * lambdafixMinus;
		float pdotm = dot(p, m);
		float A_plus = lplus2 + (lminus2 * pdotm);
		float A_minus = lplus2 - (lminus2 * pdotm);
		float epsilon = (q_plus / A_plus) - (q_minus / A_minus);

		float A = beta * epsilon;
		float B = beta * epsilonPrime;

		float gilb     = 1.0f / (1.0f + alpha * alpha);
		float mxpxmFac = gilb * (A - alpha * B);
		float pxmFac   = gilb * (B - alpha * A);

		float3 pxm      = cross(p, m);
		float3 mxpxm    = cross(m, pxm);

		tx[I] += mxpxmFac * mxpxm.x + pxmFac * pxm.x;
		ty[I] += mxpxmFac * mxpxm.y + pxmFac * pxm.y;
		tz[I] += mxpxmFac * mxpxm.z + pxmFac * pxm.z;
	}
}
