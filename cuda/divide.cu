
#include <hip/hip_runtime.h>
// dst[i] = a[i] / b[i]
extern "C" __global__ void
divide(float* __restrict__  dst, float* __restrict__  a, float* __restrict__ b, int N) {

	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

	if(i < N) {
		if((a[i] == 0) || (b[i] == 0)) {
			dst[i] = 0.0;
		} else {
			dst[i] = a[i] / b[i];
		}
	}
}
